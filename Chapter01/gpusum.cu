#include "hip/hip_runtime.h"
#include "cxtimers.h"
#include "cx.h"

__host__ __device__ inline float sinsum(float x, int terms)
{
	float x2 = x * x;
	float term = x;    // first term of series
	float sum = term;  // sum of terms so far
	for (int n = 1; n < terms; n++) {
		term *= -x2 / (2 * n * (2 * n + 1));  // build factorial
		sum += term;
	}
	return sum;
}

__global__ void gpu_sin(float *sums, int steps, int terms, float step_size)
{
	// unique thread ID
	int step = blockIdx.x * blockDim.x + threadIdx.x;
	if (step < steps) {
		float x = step_size * step;
		sums[step] = sinsum(x, terms);  // store sums
	}
}

int main(int argc, char* argv[])
{
	// get command line arguments
	int steps = (argc > 1) ? atoi(argv[1]) : 1000000;
	int terms = (argc > 2) ? atoi(argv[2]) : 1000;
	int threads = 256;
	int blocks = (steps + threads - 1) / threads;   // round up

	double pi = 3.14159265358979323;
	double step_size = pi / (steps - 1); // NB n-1
	// allocate GPU buffer and get pointer
	thrust::device_vector<float> dsums(steps);    // GPU buffer
	float *dptr = thrust::raw_pointer_cast(&dsums[0]); // get pointer
	cx::timer tim;
	gpu_sin<<<blocks, threads>>>(dptr, steps, terms, (float)step_size);
	double gpu_sum = thrust::reduce(dsums.begin(), dsums.end());
	double gpu_time = tim.lap_ms(); // get elapsed time
	// Trapezoidal Rule correction
	gpu_sum -= 0.5 * (sinsum(0.0, terms) + sinsum(pi, terms));
	gpu_sum *= step_size;
	printf("gpusum = %.10f, steps %d terms %d time %.3f ms\n",
		gpu_sum, steps, terms, gpu_time);
	return 0;
}